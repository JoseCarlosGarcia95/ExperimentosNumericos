
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

float calcular_salto(float max, unsigned long n) {
    return max / n;
}

__global__ void iteracion_gpu(float * u, unsigned long j, unsigned long m, float mu) {
    unsigned int i;

    i = threadIdx.x + blockIdx.x * blockDim.x + 1;

    u[i*m + j+1] = u[i*m + j] + mu * (u[(i-1)*m  + j] - 2*u[i*m + j] + u[(i+1)*m + j]);
}

void resolver_ecuacion_calor_explicito(float xmax, float tmax, unsigned long n, unsigned long m,
                                       float c, float (*f)(float), float (*g)(float))
{
    unsigned long i, j;
    float dx, dt, *u, mu, *u_gpu;
    hipError_t cuda_status;

    dx = calcular_salto(xmax, n);
    dt = calcular_salto(tmax, m);

    u = (float*)malloc(sizeof(float) * (n + 1) * (m + 1));
    cuda_status = hipMalloc(&u_gpu, sizeof(float) * (n + 1) * (m + 1));
    
    if (cuda_status != hipSuccess) {
        printf("hipMalloc returned error code %d\n", cuda_status);
        return;
    }
    
    for (j = 0; j <= m; j++) {
        u[j] = g(j * dt);
        u[n*m + j] = g(j * dt);
    }

    for (i = 0; i <= n; i++) {
        u[i * m] = f(i * dx);
    }

    mu = c * dt / (dx*dx);
    hipMemcpy(u_gpu, u, (n + 1) * (m + 1) * sizeof(float), hipMemcpyHostToDevice);

    for (j = 0; j < m; j++) {        
        iteracion_gpu<<<1, n-1>>>(u_gpu, j, m, mu);

        cuda_status = hipDeviceSynchronize();
		if (cuda_status != hipSuccess) {
			printf("hipDeviceSynchronize returned error code %d\n", cuda_status);
            return;
        }
    }
    
    hipMemcpy(u, u_gpu, (n + 1) * (m + 1) * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(u_gpu);    
    free(u);
}


float f_ejemplo(float x) {
    return sin(M_PI * x / 4) * (1 + 2 * cos(M_PI * x / 4));
}

float g_ejemplo(float t) {
    return 0;
}

int main(int argc, char** args) {
    int n, m;

    if (argc > 2) {
        n = atoi(args[1]);
        m = atoi(args[2]);

        resolver_ecuacion_calor_explicito(4., 0.8, n, m, 0.005, f_ejemplo, g_ejemplo);
    }
}
