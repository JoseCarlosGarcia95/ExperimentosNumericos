
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

float calcular_salto(float max, unsigned long n) {
    return max / n;
}

__global__ void iteracion_gpu(float * u, unsigned long j, unsigned long m, float lambda) {
    unsigned int i;

    i = threadIdx.x + blockIdx.x * blockDim.x + 1;

    u[i*m + j + 1] = 2*(1-lambda*lambda)*u[i*m + j] + lambda * lambda  *(u[(i+1)*m + j] + u[(i-1)*m + j]) - u[i*m + j-1];
}

void resolver_ecuacion_onda_explicito(float xmax, float tmax, unsigned long n, unsigned long m,
                                       float c, float (*f)(float), float (*g)(float))
{
    unsigned long i, j;
    float dx, dt, *u, lambda, *u_gpu;
    hipError_t cuda_status;

    dx = calcular_salto(xmax, n);
    dt = calcular_salto(tmax, m);

    u = (float*)malloc(sizeof(float) * (n + 1) * (m + 1));
    cuda_status = hipMalloc(&u_gpu, sizeof(float) * (n + 1) * (m + 1));
    
    if (cuda_status != hipSuccess) {
        printf("hipMalloc returned error code %d\n", cuda_status);
        return;
    }
    
    for (j = 1; j <= m; j++) {
        u[j] = 0;
        u[n*m + j] = 0;
    }

    u[0] = f(0);
    u[n*m] = f(xmax);

    lambda = c * dt / dx;

    for (i = 1; i < n; i++) {
        u[i*m] = f(i * dx);
        u[i*m + 1] = (1-lambda*lambda)*f(i * dx) + lambda*lambda/2 * (f((i+1)*dx) + f((i-1)*dx)) + dt * g(i * dx);
    }
    
    hipMemcpy(u_gpu, u, (n + 1) * (m + 1) * sizeof(float), hipMemcpyHostToDevice);

    for (j = 1; j < m; j++) {        
        iteracion_gpu<<<1, n-1>>>(u_gpu, j, m, lambda);

        cuda_status = hipDeviceSynchronize();
		if (cuda_status != hipSuccess) {
			printf("hipDeviceSynchronize returned error code %d\n", cuda_status);
            return;
        }
    }
    
    hipMemcpy(u, u_gpu, (n + 1) * (m + 1) * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(u_gpu);    
    free(u);
}


float f_ejemplo(float x) {
    return sin(M_PI * x / 4) * (1 + 2 * cos(M_PI * x / 4));
}

float g_ejemplo(float t) {
    return 0;
}

int main(int argc, char** args) {
    int n, m;

    if (argc > 2) {
        n = atoi(args[1]);
        m = atoi(args[2]);

        resolver_ecuacion_onda_explicito(4., 0.8, n, m, 0.005, f_ejemplo, g_ejemplo);
    }
}
